#include "hip/hip_runtime.h"
#include "imageHandling.cuh"

#include <iostream>
#include <stdio.h>

// get dimensions from a .pxa file
void imageHandling::getImageDimensions(FILE *file, int *width, int *height, int *depth) {
    fscanf(file, "%d", width);
    fscanf(file, "%d", height);
    fscanf(file, "%d", depth);
}

// load an image from a .pxa file
void imageHandling::loadImage(FILE *file, double **pixelArrays, int width, int height, int depth) {
    int offset = 0;
    while (!feof(file)) {
        for (int i = 0; i < depth; i++) {
            double tval;
            fscanf(file, "%lf", &tval);
            if (feof(file)) {
                break;
            }
            pixelArrays[i][offset] = tval;
        }
        offset += 1;
    }
    fclose(file);
}

// save an image in .pxa format
void imageHandling::saveImage(FILE *outputFile, double **pixelArrays, int width, int height, int depth) {
    fprintf(outputFile, "%d\n", width);
    fprintf(outputFile, "%d\n", height);
    fprintf(outputFile, "%d\n", depth);

    for (int j = 0; j < height; j++) {
        for (int i = 0; i < width; i++) {
            for (int k = 0; k < depth; k++) {
                int tval = (int) round(pixelArrays[k][j * width + i]);
                // clamp the values
                if (tval < 0) {
                    tval = 0;
                } else if (tval > 255) {
                    tval = 255;
                }
                fprintf(outputFile, "%d ", tval);
            }
        }
        fprintf(outputFile, "\n");
    }
    fclose(outputFile);
}

// allocate space for pixel arrays
double **imageHandling::allocateImage(int depth, int width, int height) {
    double **pixelArrays;
    hipHostMalloc(&pixelArrays, depth * sizeof(*pixelArrays));

    for (int i = 0; i < depth; i++) {
        hipHostMalloc(&(pixelArrays[i]), width * height * sizeof(double));
    }

    return pixelArrays;
}

// free pixel arrays
void imageHandling::freeImage(double **pixelArrays, int depth) {
    for (int i = 0; i < depth; i++) {
        hipHostFree(pixelArrays[i]);
    }
    hipHostFree(pixelArrays);
}