#include "imageAlgebra.cuh"

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>

void imageAlgebra::kSVD(double **pixelArrays, int width, int height, int depth, int k) {
    // set up solver handles
    hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;

    hipsolverDnCreate(&cusolverH);
    hipblasCreate(&cublasH);

    // dimensions for column major matrix (autotransposed so m > n)
    int m;
    int n;

    if (height >= width) {
        m = height;
        n = width;
    } else {
        m = width;
        n = height;
    }

    int lda = m;

    // setup host arrays
    double *A = NULL; // no need to malloc, just set to color plane pointer
    double *U = NULL; // [lda * m] n-by-m left eigenvectors
    double *VT = NULL; // [lda * n] n-by-n unitary matrix
    double *S = NULL; // [n] singular values

    hipHostMalloc(&U, lda * m * (sizeof(*U)));
    hipHostMalloc(&VT, lda * n * (sizeof(*VT)));
    hipHostMalloc(&S, n * (sizeof(*S)));

    // setup device arrays
    double *d_A = NULL;
    double *d_S = NULL;
    double *d_U = NULL;
    double *d_VT = NULL;
    int *devInfo = NULL;
    double *d_work = NULL;
    double *d_rwork = NULL;
    double *d_W = NULL;  // W = S*VT

    hipMalloc(&d_A, sizeof(double) * lda * n);
    hipMalloc(&d_S, sizeof(double) * n);
    hipMalloc(&d_U, sizeof(double) * lda * m);
    hipMalloc(&d_VT, sizeof(double) * lda * n);
    hipMalloc(&devInfo, sizeof(int));
    hipMalloc(&d_W, sizeof(double) * lda * n);

    // compute svd on each color plane
    for (int i = 0; i < depth; i++) {
        A = pixelArrays[i];

        int lwork = 0;
        const double h_one = 1;
        const double h_zero = 0;

        // copy input array to device
        hipMemcpy(d_A, A, sizeof(double) * lda * n, hipMemcpyHostToDevice);

        // query working space of SVD solver
        hipsolverDnDgesvd_bufferSize(cusolverH, m, n, &lwork);
        hipMalloc(&d_work, sizeof(double) * lwork);

        // compute SVD
        signed char jobu = 'A'; // all m columns of U
        signed char jobvt = 'A'; // all n columns of VT
        hipsolverDnDgesvd(
                cusolverH,
                jobu,
                jobvt,
                m,
                n,
                d_A,
                lda,
                d_S,
                d_U,
                lda,  // ldu
                d_VT,
                lda, // ldvt,
                d_work,
                lwork,
                d_rwork,
                devInfo);

        hipDeviceSynchronize();

        hipFree(d_work);

        // copy singular value results back
        hipMemcpy(S, d_S, sizeof(double) * n, hipMemcpyDeviceToHost);

        // save only data at indicies where singular value is in top k
        for (int j = 0; j < n; j++) {
            if (j >= k) {
                S[j] = 0.0;
            }
        }
        hipMemcpy(d_S, S, sizeof(double) * n, hipMemcpyHostToDevice);

        // W = S*VT
        hipblasDdgmm(
                cublasH,
                HIPBLAS_SIDE_LEFT,
                n,
                n,
                d_VT,
                lda,
                d_S,
                1,
                d_W,
                lda);

        hipDeviceSynchronize();

        // A = U*W
        hipblasDgemm(
                cublasH,
                HIPBLAS_OP_N, // U
                HIPBLAS_OP_N, // W
                m, // number of rows of A
                n, // number of columns of A
                n, // number of columns of U
                &h_one, // host pointer
                d_U, // U
                lda,
                d_W, // W
                lda,
                &h_zero, // host pointer
                d_A,
                lda);

        hipDeviceSynchronize();

        // copy results back
        hipMemcpy(A, d_A, sizeof(double) * lda * n, hipMemcpyDeviceToHost);
    }

    // free resources
    hipFree(d_A);
    hipFree(d_S);
    hipFree(d_U);
    hipFree(d_VT);
    hipFree(d_W);

    hipFree(devInfo);
    hipFree(d_rwork);

    hipblasDestroy(cublasH);
    hipsolverDnDestroy(cusolverH);

    hipHostFree(U);
    hipHostFree(S);
    hipHostFree(VT);
}
